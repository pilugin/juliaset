#include "hip/hip_runtime.h"
#include "julia.cu.h"

namespace {

struct hipComplex
{
    float r;
    float i;

    __device__ hipComplex(float rr = 0.0, float ii = 0.0) : r{rr}, i{ii} { ; }

    __device__ float magnitude2() const
    {
        return r*r + i*i;
    }

    __device__ hipComplex operator+(const hipComplex& other) const
    {
        return hipComplex{r + other.r, i + other.i};
    }

    __device__ hipComplex operator*(const hipComplex& other) const
    {
        return hipComplex{r*other.r - i*other.i, i*other.r + r*other.i};
    }
};

constexpr float DIM = 1000.;

__device__ int julia(int x, int y, float scale, size_t maxIter)
{
    float jx = scale * (DIM/2. - x)/(DIM/2.);
    float jy = scale * (DIM/2. - y)/(DIM/2.);

    hipComplex c{-0.8, 0.156}; // -0.8; 0.156
    hipComplex a{jx, jy};
    int i=0;
    for (; i<maxIter; ++i)
    {
        a = a*a + c;
        if (a.magnitude2() > 1000)
        {
            break;
        }
    }
    return i;
}

__global__ void juliaKernel(uchar4* ptr, int w, int h, float scale, const uchar4* gradient, size_t gradientSize)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y*w;

    if (x >= w || y >= h)
    {
        return;
    }

    uchar4& pixel = ptr[offset];

    const size_t maxIter = gradientSize * 2;

    int juliaValue = julia(x, y, scale, maxIter);

    const float c = static_cast<float>(juliaValue) / static_cast<float>(maxIter);

    pixel = gradient[ static_cast<int>(static_cast<float>(gradientSize - 1) * c) ];
}

} // namespace

void renderJuliaSet(void* devPtr, int w, int h, double scaleFactor, const void* gradient, size_t gradientSize)
{
    const unsigned int blockSize = 32;

    dim3 grid{(w + blockSize-1)/blockSize, (h + blockSize - 1)/blockSize};
    dim3 block{blockSize, blockSize};

    juliaKernel<<<grid, block>>>
                            (static_cast<uchar4*>(devPtr), w, h, scaleFactor,
                             static_cast<const uchar4*>(gradient), gradientSize);
}
